
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

//#include <GL/glew.h>
//#include <GL/freeglut.h>
 
#define SRAND_VALUE 1985
#define BLOCK_SIZE 16
 
struct timespec begin, end;
double time_spent;

__global__ void ghostRows(int dim, int *grid)
{
    // We want id ∈ [1,dim]
    int id = blockDim.x * blockIdx.x + threadIdx.x + 1;
 
    if (id <= dim)
    {
        //Copy first real row to bottom ghost row
        grid[(dim+2)*(dim+1)+id] = grid[(dim+2)+id];
        //Copy last real row to top ghost row
        grid[id] = grid[(dim+2)*dim + id]; 
    }
}
 
__global__ void ghostCols(int dim, int *grid)
{
    // We want id ∈ [0,dim+1]
    int id = blockDim.x * blockIdx.x + threadIdx.x;
 
    if (id <= dim+1)
    {
        //Copy first real column to right most ghost column
        grid[id*(dim+2)+dim+1] = grid[id*(dim+2)+1];
       //Copy last real column to left most ghost column
        grid[id*(dim+2)] = grid[id*(dim+2) + dim];
    }
}
 
__global__ void GOL(int dim, int *grid, int *newGrid)
{
    // We want id ∈ [1,dim]
    int iy = blockDim.y * blockIdx.y + threadIdx.y + 1;
    int ix = blockDim.x * blockIdx.x + threadIdx.x + 1;
    int id = iy * (dim+2) + ix;
 
    int numNeighbors;
 
    if (iy <= dim && ix <= dim) {
 
        // Get the number of neighbors for a given grid point
        numNeighbors = grid[id+(dim+2)] + grid[id-(dim+2)] //upper lower
                     + grid[id+1] + grid[id-1]             //right left
                     + grid[id+(dim+3)] + grid[id-(dim+3)] //diagonals
                     + grid[id-(dim+1)] + grid[id+(dim+1)];
 
        int cell = grid[id];
        // Here we have explicitly all of the game rules
        if (cell == 1 && numNeighbors < 2)
            newGrid[id] = 0;
        else if (cell == 1 && (numNeighbors == 2 || numNeighbors == 3))
            newGrid[id] = 1;
        else if (cell == 1 && numNeighbors > 3)
            newGrid[id] = 0;
        else if (cell == 0 && numNeighbors == 3)
            newGrid[id] = 1;
        else
            newGrid[id] = cell;
    }
}
 
int main(int argc, char* argv[])
{
    int i,j,iter;
    int *h_grid; //Grid on host
    int *d_grid; //Grid on device
    int* d_newGrid; //Second grid used on device only
    int* d_tmpGrid; //tmp grid pointer used to switch between grid and newGrid
 
    int dim = 128; //Linear dimension of our grid - not counting ghost cells
    int maxIter = 10000; //Number of game steps
 
    size_t bytes = sizeof(int)*(dim+2)*(dim+2);
    // Allocate host Grid used for initial setup and read back from device
    h_grid = (int*)malloc(bytes);
 
    // Allocate device grids
    hipMalloc(&d_grid, bytes);
    hipMalloc(&d_newGrid, bytes);
 
    // Assign initial population randomly
    srand(SRAND_VALUE);
    for(i = 1; i<=dim; i++) {
        for(j = 1; j<=dim; j++) {
            h_grid[i*(dim+2)+j] = rand() % 2;
        }
    }
 
    // Copy over initial game grid (Dim-1 threads)
    hipMemcpy(d_grid, h_grid, bytes, hipMemcpyHostToDevice);
 
    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    int linGrid = (int)ceil(dim/(float)BLOCK_SIZE);
    dim3 gridSize(linGrid,linGrid);
 
    dim3 cpyBlockSize(BLOCK_SIZE);
    dim3 cpyGridRowsGridSize((int)ceil(dim/(float)cpyBlockSize.x));
    dim3 cpyGridColsGridSize((int)ceil((dim+2)/(float)cpyBlockSize.x));
 
		//begin timing
		clock_gettime(CLOCK_MONOTONIC, &begin);

    // Main game loop
    for (iter = 0; iter<maxIter; iter++) {
 
        ghostRows<<<cpyGridRowsGridSize, cpyBlockSize>>>(dim, d_grid);
        ghostCols<<<cpyGridColsGridSize, cpyBlockSize>>>(dim, d_grid);
        GOL<<<blockSize, gridSize>>>(dim, d_grid, d_newGrid);
 
        // Swap our grids and iterate again
        d_tmpGrid = d_grid;
        d_grid = d_newGrid;
        d_newGrid = d_tmpGrid;
    }//iter loop
 
		clock_gettime(CLOCK_MONOTONIC, &end);
		time_spent = (end.tv_sec - begin.tv_sec);
		time_spent = time_spent + (end.tv_nsec - begin.tv_nsec) / 1000000000.0;

		printf("\nTime taken is: %f\n", time_spent);

    // Copy back results and sum
    hipMemcpy(h_grid, d_grid, bytes, hipMemcpyDeviceToHost);
 
    // Sum up alive cells and print results
    int total = 0;
    for (i = 1; i<=dim; i++)
    {
        for (j = 1; j<=dim; j++)
        {
            total += h_grid[i*(dim+2)+j];
        }
        //printf("\n");
    }
    printf("Total Alive: %d\n", total);
   
    // Release memory
    hipFree(d_grid);
    hipFree(d_newGrid);
    free(h_grid);
 
    return 0;
}
